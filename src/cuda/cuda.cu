#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

// Struct to represent the Gaussian Matrix
struct GaussianMatrix {
    int size;
    double *mat;
};

// Initialize the matrix based on user input
void initialize_matrix(GaussianMatrix& matrix) {
    // Get the matrix size
    cin >> matrix.size;

    // Allocate memory for the matrix
    matrix.mat = new double[matrix.size * matrix.size * 2];
}

// Function to get the input matrix from user input and init identity
void input_matrix(GaussianMatrix& matrix) {
    // Get the user input
    for (int i = 0; i < matrix.size; ++i) {
        for (int j = 0; j < matrix.size; ++j) {
            cin >> matrix.mat[i * matrix.size * 2 + j];
        }

        // Initialize right-hand side to identity matrix
        for (int j = matrix.size; j < matrix.size * 2; ++j) {
            matrix.mat[i * matrix.size * 2 + j] = (j == i + matrix.size) ? 1 : 0;
        }
    }
}

// Free the memory allocated for the matrix
void deallocate_matrix(GaussianMatrix& matrix) {
    delete[] matrix.mat;
}

// Perform gaussian elimination
__global__ void gaussian_elimination_kernel(double *matrix, int size, int pivot) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size && idx != pivot) {
        double ratio = matrix[idx * size * 2 + pivot] / matrix[pivot * size * 2 + pivot];
        for (int j = pivot; j < 2 * size; ++j) {
            matrix[idx * size * 2 + j] -= matrix[pivot * size * 2 + j] * ratio;
        }
    }
}

// Reduce the matrix to a unit matrix
__global__ void reduce_to_unit_kernel(double *matrix, int size, int pivot) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size && idx == pivot) {
        double pivot_value = matrix[idx * size * 2 + pivot];
        if (pivot_value != 0.0) {
            for (int j = idx; j < 2 * size; ++j) {
                matrix[idx * size * 2 + j] /= pivot_value;
            }
        }
    }
}

// Function to perform the matrix inversion in CUDA
void matrix_inversion_cuda(GaussianMatrix& matrix) {
    // Define and initiate value of the GPU matrix
    int size = matrix.size;
    double *d_matrix;

    // Allocate GPU memory for the matrix and copy the value
    hipError_t err = hipMalloc(&d_matrix, size * size * 2 * sizeof(double));
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__,__LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_matrix, matrix.mat, size * size * 2 * sizeof(double), hipMemcpyHostToDevice);

    // Define the value of thread per block and block per grid
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Doing the paralel process on every pivot
    for (int pivot = 0; pivot < size; ++pivot) {
        // Do gaussian elimination
        gaussian_elimination_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_matrix, size, pivot);
        hipDeviceSynchronize();

        // Reduce the matrix to a unit matrix
        reduce_to_unit_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_matrix, size, pivot);
        hipDeviceSynchronize();
    }

    // Copy the result back to CPU memory
    hipMemcpy(matrix.mat, d_matrix, size * size * 2 * sizeof(double), hipMemcpyDeviceToHost);
    
    // Free the GPU memory
    hipFree(d_matrix);
}

// Function to print the resulting matrix
void print_result(GaussianMatrix& matrix) {
    cout << matrix.size << endl;
    for (int i = 0; i < matrix.size; ++i) {
        for (int j = matrix.size; j < 2 * matrix.size; ++j) {
            cout << matrix.mat[i * matrix.size * 2 + j] << " ";
        }
        cout << endl;
    }
}

int main() {
    // Initaite object
    GaussianMatrix matrix;
    
    // Initialize the matrix based on user input
    initialize_matrix(matrix);
    
    // Get the input matrix from user input
    input_matrix(matrix);

    // Perform matrix inversion in CUDA
    matrix_inversion_cuda(matrix);
    
    // Print the result
    print_result(matrix);

    // Free the memory allocated for the matrix
    deallocate_matrix(matrix);

    return 0;
}
